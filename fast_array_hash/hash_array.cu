#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <vector>


/* Hashing function

Uses a simple LSH (Locality Sensitive Hashing) function to hash an integer
value.

@param value: The value to hash
@param seed: A seed value to use in the hash function

@return: The hashed value
*/
__device__ auto lsh_hash(auto value, const auto seed) {
    value ^= seed;
    value *= 0x3243f6a8885a308d;
    value ^= value >> 15;
    value *= 0x1111111111111111;
    value ^= value >> 15;

    return value;
}


/* Pairing function

Uses the Szudzik pairing function to combine two signed 32 bit integers
into a single unsigned 64 bit integer.

@param a: The first integer
@param b: The second integer

@return: The combined integer
*/
__device__ u_int64_t szudzikPairing(int32_t a, int32_t b) {
    uint64_t A = a >= 0 ? 2 * static_cast<uint64_t>(a) : -2 * static_cast<uint64_t>(a) - 1;
    uint64_t B = b >= 0 ? 2 * static_cast<uint64_t>(b) : -2 * static_cast<uint64_t>(b) - 1;
    uint64_t C = A >= B ? A * A + A + B : A + B * B;
    return C;
}


int main() {
    const auto array_size = 1 << 20;  // 1 million elements
    const auto seed = 42;

    // Allocate memory on host
    std::vector<unsigned int> host_input(array_size);
    unsigned int host_output;

    // Initialize input array
    for (auto i = 0; i < array_size; i++) {
        host_input[i] = i;
    }

    // Allocate memory on device
    unsigned int* device_input;
    unsigned int* device_output;
    hipMalloc((void**)&device_input, array_size * sizeof(unsigned int));
    hipMalloc((void**)&device_output, sizeof(unsigned int));

    // Copy input data from host to device
    hipMemcpy(device_input, host_input.data(), array_size * sizeof(unsigned int), hipMemcpyHostToDevice);

    // Launch kernel
    const auto block_size = 256;
    const auto grid_size = (array_size + block_size - 1) / block_size;
    lsh_hash_kernel<<<grid_size, block_size>>>(device_input, device_output, array_size, seed);

    // Copy output data from device to host
    hipMemcpy(&host_output, device_output, sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Print the hash of the array
    std::cout << "Output: " << host_output << std::endl;

    // Free device memory
    hipFree(device_input);
    hipFree(device_output);

    return 0;
}